#include "hip/hip_runtime.h"
#foreach( $degs in $degrees )

// P(X)/Q(X) = a_0 + a_1*X + a_2*X^2 + ... + a_n*X^n / eps + |b_0 + b_1*X + b_2*X^2 + ... + b_n*X^n|
// eps = 0.0000001

#set( $degs_a = $degs[0] )
#set( $degs_b = $degs[1] )
#set( $coefs_a = $degs_a )
#set( $coefs_b = $degs_b - 1 )
#set( $a_counts = $coefs_a + 1 )
#set( $b_counts = $coefs_b + 1 )
#set( $max_x = $degs[2] )

template <typename scalar_t>
__global__ void pau_cuda_forwardC_kernel_$degs[0]_$degs[1]( const scalar_t* __restrict__ x, const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b, scalar_t* __restrict__ result, size_t x_size) {

    #foreach( $idx in [0..$coefs_a] )
    scalar_t a_$idx = a[$idx];
    #end

    #foreach( $idx in [0..$coefs_b] )
    scalar_t b_$idx = b[$idx];
    #end

    scalar_t eps = scalar_t(0.0000001);

    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
        index < x_size;
        index += blockDim.x * gridDim.x){

        scalar_t xp1 = x[index];

        #foreach( $idx in [2..$max_x] )#set( $value = $idx - 1 )
        scalar_t xp$idx = xp$value * xp1;
        #end

        scalar_t P = a_0
        #foreach( $idx in [1..$coefs_a] )
        + a_$idx * xp$idx
        #end
        ;

        scalar_t Q = eps + abs(b_0
        #foreach( $idx in [1..$coefs_b] )
        + b_$idx * xp$idx
        #end
        );

        result[index] = P/Q;
    }
}


at::Tensor pau_cuda_forwardC_$degs[0]_$degs[1](torch::Tensor x, torch::Tensor n, torch::Tensor d){
    auto result = at::empty_like(x);
    const auto x_size = x.numel();

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = (x_size + blockSize - 1) / blockSize;

    AT_DISPATCH_FLOATING_TYPES(x.type(), "pau_cuda_forwardC_$degs[0]_$degs[1]", ([&] {
    pau_cuda_forwardC_kernel_$degs[0]_$degs[1]<scalar_t>
        <<<numBlocks, blockSize>>>(
            x.data<scalar_t>(),
            n.data<scalar_t>(),
            d.data<scalar_t>(),
            result.data<scalar_t>(),
            x_size);
        }));

    return result;
}




//P(X) = a_0 + a_1*X + a_2*X^2 ...
//Q(X) = eps + |A(X)|
//R(X) = a_1 + 2*a_2*X + 3*a_3*X ...
//A(X) = b_0 + b_1*X + b_2*X^2 + b_3*X^3
//S(X) = sign(A(X)) * ( b_1 + 2*b_2*X + 3*b_3*X^2 ...)
//dF/dx = (-P(X)/Q(X)^2)*S(X) + R(X)/Q(X)
//dF/da_i = x^i/Q(X), i \in {0,$degs[0]}
//dF/db_i = (-P(X)/Q(X)^2) * sign(A(X)) * X^i , i \in {0,$degs[1]}
//eps = 0.0000001

template <typename scalar_t>
__global__ void pau_cuda_backwardC_kernel_$degs[0]_$degs[1](
    const scalar_t* __restrict__ grad_output,
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ d_x,
    double* __restrict__ d_a,
    double* __restrict__ d_b,
    size_t x_size) {

    __shared__ double sda[$a_counts];
    __shared__ double sdb[$b_counts];

    scalar_t eps = scalar_t(0.0000001);

    if( threadIdx.x == 0){
        #foreach( $idx in [0..$coefs_a] )
        sda[$idx] = 0;
        #end
        #foreach( $idx in [0..$coefs_b] )
        sdb[$idx] = 0;
        #end
    }

    __syncthreads();
    #foreach( $idx in [0..$coefs_a] )
    scalar_t d_a$idx = 0;
    scalar_t a_$idx = a[$idx];
    #end

    #foreach( $idx in [0..$coefs_b] )
    scalar_t d_b$idx = 0;
    scalar_t b_$idx = b[$idx];
    #end

    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < x_size;
         index += blockDim.x * gridDim.x)
      {
        scalar_t xp1 = x[index];

        #foreach( $idx in [2..$max_x] )#set( $value = $idx - 1 )
        scalar_t xp$idx = xp$value * xp1;
        #end

        scalar_t P = a_0
        #foreach( $idx in [1..$coefs_a] )
        + a_$idx*xp$idx
        #end
        ;

        scalar_t A = b_0
        #foreach( $idx in [1..$coefs_b] )
        + b_$idx * xp$idx
        #end
        ;

        scalar_t Q = eps + abs(A);

        scalar_t R = a_1
        #foreach( $idx in [2..$coefs_a] )#set( $value = $idx - 1 )
        + scalar_t($idx.0) * a_$idx * xp$value
        #end
        ;

        scalar_t S = copysign( scalar_t(1.0), A ) * (b_1

        #foreach( $idx in [2..$coefs_b] )#set( $value = $idx - 1 )
        + scalar_t($idx.0) * b_$idx * xp$value
        #end
         );

        scalar_t mpq2 = -P/(Q*Q);

        scalar_t grad_o = grad_output[index];

        scalar_t d_i_x = (R/Q + S*mpq2);
        d_x[index] = d_i_x * grad_o;

        scalar_t d_i_b0 = mpq2 * copysign( scalar_t(1.0), A );
        d_b0 += d_i_b0 * grad_o;

        #foreach( $idx in [1..$coefs_b] )
        scalar_t d_i_b$idx = mpq2 * copysign( scalar_t(1.0), A ) * xp$idx;
        d_b$idx += d_i_b$idx * grad_o;
        #end

        scalar_t d_i_a0 = scalar_t(1.0)/Q;
        d_a0 += d_i_a0 * grad_o;

        #foreach( $idx in [1..$coefs_a] )#set( $value = $idx - 1 )
        scalar_t d_i_a$idx  = xp$idx/Q;
        d_a$idx += d_i_a$idx * grad_o;
        #end
    }

    #foreach( $idx in [0..$coefs_a] )
    atomicAdd(&sda[$idx], d_a$idx);
    #end
    #foreach( $idx in [0..$coefs_b] )
    atomicAdd(&sdb[$idx], d_b$idx);
    #end

    __syncthreads();

    if( threadIdx.x == 0){
        #foreach( $idx in [0..$coefs_a] )
        atomicAdd(&d_a[$idx], sda[$idx]);
        #end
        #foreach( $idx in [0..$coefs_b] )
        atomicAdd(&d_b[$idx], sdb[$idx]);
        #end
    }
}




std::vector<torch::Tensor> pau_cuda_backwardC_$degs[0]_$degs[1](torch::Tensor grad_output, torch::Tensor x, torch::Tensor n, torch::Tensor d){
    const auto x_size = x.numel();
    auto d_x = at::empty_like(x);
    auto d_n = at::zeros_like(n).toType(at::kDouble);
    auto d_d = at::zeros_like(d).toType(at::kDouble);

    int blockSize = THREADS_PER_BLOCK;

    AT_DISPATCH_FLOATING_TYPES(x.type(), "pau_cuda_backwardC_$degs[0]_$degs[1]", ([&] {
    pau_cuda_backwardC_kernel_$degs[0]_$degs[1]<scalar_t>
        <<<16, blockSize>>>(
            grad_output.data<scalar_t>(),
            x.data<scalar_t>(),
            n.data<scalar_t>(),
            d.data<scalar_t>(),
            d_x.data<scalar_t>(),
            d_n.data<double>(),
            d_d.data<double>(),
            x_size);
    }));

    return {d_x, d_n.toType(at::kFloat), d_d.toType(at::kFloat)};
}

#end








